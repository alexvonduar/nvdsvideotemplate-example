#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "nv12_to_rgb_kernel.hpp"

#define checkRuntime(call)  check_runtime(call, #call, __LINE__, __FILE__)
#define half2short(h)   (*(unsigned short*)&h)

typedef unsigned char uint8_t;

template<typename _T>struct AsUnion4{};
template<>struct AsUnion4<uint8_t>{typedef uchar4  type;};
template<>struct AsUnion4<float>  {typedef float4  type;};
template<>struct AsUnion4<__half> {typedef ushort4 type;};

template<typename _T>struct AsUnion3{};
template<>struct AsUnion3<uint8_t>{typedef uchar3  type;};
template<>struct AsUnion3<float>  {typedef float3  type;};
template<>struct AsUnion3<__half> {typedef ushort3 type;};

template<DataType _T>struct AsPODType{};
template<>struct AsPODType<DataType::Uint8>   {typedef uint8_t type;};
template<>struct AsPODType<DataType::Float32> {typedef float   type;};
template<>struct AsPODType<DataType::Float16> {typedef __half  type;};

enum class Parallel : unsigned int{
    None        = 0,
    SinglePixel = 1,
    FourPixel   = 2
};

static __device__ __forceinline__ uchar4 make4(uint8_t v0, uint8_t v1, uint8_t v2, uint8_t v3){return make_uchar4(v0, v1, v2, v3);}
static __device__ __forceinline__ float4 make4(float v0, float v1, float v2, float v3){return make_float4(v0, v1, v2, v3);}
static __device__ __forceinline__ ushort4 make4(__half v0, __half v1, __half v2, __half v3){return make_ushort4(half2short(v0), half2short(v1), half2short(v2), half2short(v3)); }

static __device__ __forceinline__ uchar3 make3(uint8_t v0, uint8_t v1, uint8_t v2){return make_uchar3(v0, v1, v2);}
static __device__ __forceinline__ float3 make3(float v0, float v1, float v2){return make_float3(v0, v1, v2);}
static __device__ __forceinline__ ushort3 make3(__half v0, __half v1, __half v2){return make_ushort3(half2short(v0), half2short(v1), half2short(v2)); }

#define INTER_RESIZE_COEF_BITS 11
#define INTER_RESIZE_COEF_SCALE (1 << INTER_RESIZE_COEF_BITS)
#define CAST_BITS (INTER_RESIZE_COEF_BITS << 1)

template<typename _T>
static __forceinline__ __device__ _T limit(_T value, _T low, _T high){
    return value < low ? low : (value > high ? high : value);
}

template<typename _T>
static __host__ __device__ __forceinline__ uint8_t u8cast(_T value){
    return value < 0 ? 0 : (value >= 255 ? 255 : value);
}

template<typename _T>
static __host__ __device__ __forceinline__ _T fpcast(_T value){
    return value < 0 ? 0 : (value >= 255 ? 255 : value);
}

static bool __inline__ check_runtime(hipError_t e, const char* call, int line, const char *file){
    if (e != hipSuccess) {
        fprintf(stderr, "CUDA Runtime error %s # %s, code = %s [ %d ] in file %s:%d\n", call, hipGetErrorString(e), hipGetErrorName(e), e, file, line);
        return false;
    }
    return true;
}

template<typename _DataType, PixelLayout _Layout, Parallel parallel>
struct DataLayoutInvoker{};

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// NHWC RGB
template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NHWC_RGB, Parallel::SinglePixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r, _DataType g, _DataType b, int ib, int x, int y, int width, int stride, int height){

        _DataType* p = pdst + (ib * height + y) * stride + x * 3;
        p[0] = r; p[1] = g; p[2] = b;
    }
};

template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NHWC_RGB, Parallel::FourPixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r[4], _DataType g[4], _DataType b[4], int ib, int x, int y, int width, int stride, int height){
        
        _DataType* p0 = pdst + (ib * height + y) * stride + (x+0) * 3;
        _DataType* p1 = pdst + (ib * height + y) * stride + (x+1) * 3;
        _DataType* p2 = pdst + (ib * height + y) * stride + (x+2) * 3;
        _DataType* p3 = pdst + (ib * height + y) * stride + (x+3) * 3;
        *(typename AsUnion3<_DataType>::type*)p0 = make3(r[0], g[0], b[0]);
        *(typename AsUnion3<_DataType>::type*)p1 = make3(r[1], g[1], b[1]);
        *(typename AsUnion3<_DataType>::type*)p2 = make3(r[2], g[2], b[2]);
        *(typename AsUnion3<_DataType>::type*)p3 = make3(r[3], g[3], b[3]);
    }
};

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// NHWC BGR
template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NHWC_BGR, Parallel::SinglePixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r, _DataType g, _DataType b, int ib, int x, int y, int width, int stride, int height){

        _DataType* p = pdst + (ib * height + y) * stride + x * 3;
        p[0] = b; p[1] = g; p[2] = r;
    }
};

template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NHWC_BGR, Parallel::FourPixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r[4], _DataType g[4], _DataType b[4], int ib, int x, int y, int width, int stride, int height){

        _DataType* p0 = pdst + (ib * height + y) * stride + (x+0) * 3;
        _DataType* p1 = pdst + (ib * height + y) * stride + (x+1) * 3;
        _DataType* p2 = pdst + (ib * height + y) * stride + (x+2) * 3;
        _DataType* p3 = pdst + (ib * height + y) * stride + (x+3) * 3;
        *(typename AsUnion3<_DataType>::type*)p0 = make3(b[0], g[0], r[0]);
        *(typename AsUnion3<_DataType>::type*)p1 = make3(b[1], g[1], r[1]);
        *(typename AsUnion3<_DataType>::type*)p2 = make3(b[2], g[2], r[2]);
        *(typename AsUnion3<_DataType>::type*)p3 = make3(b[3], g[3], r[3]);
    }
};

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// NCHW RGB
template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW_RGB, Parallel::SinglePixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r, _DataType g, _DataType b, int ib, int x, int y, int width, int stride, int height){

        *(pdst + (((ib * 3 + 0) * height + y) * width + x)) = r;
        *(pdst + (((ib * 3 + 1) * height + y) * width + x)) = g;
        *(pdst + (((ib * 3 + 2) * height + y) * width + x)) = b;
    }
};

template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW_RGB, Parallel::FourPixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r[4], _DataType g[4], _DataType b[4], int ib, int x, int y, int width, int stride, int height){

        *(typename AsUnion4<_DataType>::type*)(pdst + (((ib * 3 + 0) * height + y) * width + x)) = make4(r[0], r[1], r[2], r[3]);
        *(typename AsUnion4<_DataType>::type*)(pdst + (((ib * 3 + 1) * height + y) * width + x)) = make4(g[0], g[1], g[2], g[3]);
        *(typename AsUnion4<_DataType>::type*)(pdst + (((ib * 3 + 2) * height + y) * width + x)) = make4(b[0], b[1], b[2], b[3]);
    }
};

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// NCHW BGR
template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW_BGR, Parallel::SinglePixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r, _DataType g, _DataType b, int ib, int x, int y, int width, int stride, int height){

        *(pdst + (((ib * 3 + 0) * height + y) * width + x)) = b;
        *(pdst + (((ib * 3 + 1) * height + y) * width + x)) = g;
        *(pdst + (((ib * 3 + 2) * height + y) * width + x)) = r;
    }
};

template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW_BGR, Parallel::FourPixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r[4], _DataType g[4], _DataType b[4], int ib, int x, int y, int width, int stride, int height){

        *(typename AsUnion4<_DataType>::type*)(pdst + (((ib * 3 + 0) * height + y) * width + x)) = make4(b[0], b[1], b[2], b[3]);
        *(typename AsUnion4<_DataType>::type*)(pdst + (((ib * 3 + 1) * height + y) * width + x)) = make4(g[0], g[1], g[2], g[3]);
        *(typename AsUnion4<_DataType>::type*)(pdst + (((ib * 3 + 2) * height + y) * width + x)) = make4(r[0], r[1], r[2], r[3]);
    }
};

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// NCHW16 RGB
template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW16_RGB, Parallel::SinglePixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r, _DataType g, _DataType b, int ib, int x, int y, int width, int stride, int height){

        _DataType* p = pdst + ((ib * height + y) * width + x) * 16;
        p[0] = r; p[1] = g; p[2] = b;
    }
};

template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW16_RGB, Parallel::FourPixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r[4], _DataType g[4], _DataType b[4], int ib, int x, int y, int width, int stride, int height){

        _DataType* p0 = pdst + ((ib * height + y) * width + x+0) * 16;
        _DataType* p1 = pdst + ((ib * height + y) * width + x+1) * 16;
        _DataType* p2 = pdst + ((ib * height + y) * width + x+2) * 16;
        _DataType* p3 = pdst + ((ib * height + y) * width + x+3) * 16;
        *(typename AsUnion3<_DataType>::type*)p0 = make3(r[0], g[0], b[0]);
        *(typename AsUnion3<_DataType>::type*)p1 = make3(r[1], g[1], b[1]);
        *(typename AsUnion3<_DataType>::type*)p2 = make3(r[2], g[2], b[2]);
        *(typename AsUnion3<_DataType>::type*)p3 = make3(r[3], g[3], b[3]);
    }
};

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////// NCHW16 BGR
template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW16_BGR, Parallel::SinglePixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r, _DataType g, _DataType b, int ib, int x, int y, int width, int stride, int height){

        _DataType* p = pdst + ((ib * height + y) * width + x) * 16;
        p[0] = b; p[1] = g; p[2] = r;
    }
};

template<typename _DataType>
struct DataLayoutInvoker<_DataType, PixelLayout::NCHW16_BGR, Parallel::FourPixel>{
    static __device__ __forceinline__ void call(_DataType* pdst, _DataType r[4], _DataType g[4], _DataType b[4], int ib, int x, int y, int width, int stride, int height){

        _DataType* p0 = pdst + ((ib * height + y) * width + x+0) * 16;
        _DataType* p1 = pdst + ((ib * height + y) * width + x+1) * 16;
        _DataType* p2 = pdst + ((ib * height + y) * width + x+2) * 16;
        _DataType* p3 = pdst + ((ib * height + y) * width + x+3) * 16;
        *(typename AsUnion3<_DataType>::type*)p0 = make3(b[0], g[0], r[0]);
        *(typename AsUnion3<_DataType>::type*)p1 = make3(b[1], g[1], r[1]);
        *(typename AsUnion3<_DataType>::type*)p2 = make3(b[2], g[2], r[2]);
        *(typename AsUnion3<_DataType>::type*)p3 = make3(b[3], g[3], r[3]);
    }
};
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ unsigned int __forceinline__ round_down2(unsigned int num){
    return num & (~1);
}

template<typename _T>
static __device__ void __forceinline__ scale_rgb(
    uint8_t r0, uint8_t g0, uint8_t b0, _T& r, _T& g, _T& b,
    float mean0, float mean1, float mean2, float scale0, float scale1, float scale2
){
    r = (r0 - mean0) * scale0;
    g = (g0 - mean1) * scale1;
    b = (b0 - mean2) * scale2;
}

static __device__ void __forceinline__ yuv2rgb(
    uint8_t y, uint8_t u, uint8_t v, uint8_t& r, uint8_t& g, uint8_t& b
){
    int iyval = 1220542*(y - 16);
    r = u8cast((iyval + 1673527*(v - 128)                      + (1 << 19)) >> 20);
    g = u8cast((iyval - 852492*(v - 128) - 409993*(u - 128)    + (1 << 19)) >> 20);
    b = u8cast((iyval                      + 2116026*(u - 128) + (1 << 19)) >> 20);
}

template<NV12Format nv12_format>
static __device__ void __forceinline__ load_nv12_pixel(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
);

// BL sample pixel implmentation
template<>
__device__ void __forceinline__ load_nv12_pixel<NV12Format::BlockLinear>(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uint8_t yv = tex2D<uint8_t>((hipTextureObject_t)luma,   x,          y    );
    uchar2 uv = tex2D<uchar2>((hipTextureObject_t)chroma, x / 2, y / 2);
    yuv2rgb(yv, uv.x, uv.y, r, g, b);
}

// PL sample pixel implmentation
template<>
__device__ void __forceinline__ load_nv12_pixel<NV12Format::PitchLinear>(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uint8_t yv = *((const unsigned char*)luma + y * stride + x);
    uint8_t uv = *((const unsigned char*)chroma + (y / 2) * stride + down_x + 0);
    uint8_t vv = *((const unsigned char*)chroma + (y / 2) * stride + down_x + 1);
    yuv2rgb(yv, uv, vv, r, g, b);
}

template<typename DType, NV12Format nv12_format, Interpolation interp>
struct SamplePixel{};

// BL sample pixel implmentation
template<typename DType, NV12Format format>
struct SamplePixel<DType, format, Interpolation::Nearest>{
    static __device__ void __forceinline__ call(
        const void* luma, const void* chroma, 
        int x, int y, float sx, float sy, int ybatch_offset, int width, int stride, int height,
        uint8_t& r, uint8_t& g, uint8_t& b
    ){
        // In some cases, the floating point precision will lead to miscalculation of the value, 
        // making the result not exactly match with opencv, 
        // so here you need to add eps as precision compensation
        //
        // A special case is when the input is 3840 and the output is 446, x = 223:
        // const int src_x_double = 223.0  * (3840.0  / 446.0);            // -> 1920
        // const int src_x_float  = 223.0f * (3840.0f / 446.0f);           // -> 1919
        // const int src_x_float  = 223.0f * (3840.0f / 446.0f) + 1e-5;    // -> 1920
        //
        // !!! If you want to use the double for sx/sy, you'll get a 2x speed drop
        const float eps = 1e-5;
        int ix = x * sx + eps;
        int iy = y * sy + eps + ybatch_offset;
        load_nv12_pixel<format>(luma, chroma, ix, iy, round_down2(ix), width, stride, r, g, b);
    }
};

template<typename DType, NV12Format format>
struct SamplePixel<DType, format, Interpolation::Bilinear>{
    static __device__ void __forceinline__ call(
        const void* luma, const void* chroma, 
        int x, int y, float sx, float sy, int ybatch_offset, int width, int stride, int height,
        uint8_t& r, uint8_t& g, uint8_t& b
    ){
        uint8_t r0[4], g0[4], b0[4];
        float src_x = (x + 0.5f) * sx - 0.5f;
        float src_y = (y + 0.5f) * sy - 0.5f;
        int y_low  = floorf(src_y);
        int x_low  = floorf(src_x);
        int y_high = limit(y_low + 1, 0, height - 1);
        int x_high = limit(x_low + 1, 0, width - 1);
        y_low = limit(y_low, 0, height - 1);
        x_low = limit(x_low, 0, width - 1);

        int ly = rint((src_y - y_low) * INTER_RESIZE_COEF_SCALE);
        int lx = rint((src_x - x_low) * INTER_RESIZE_COEF_SCALE);
        int hy = INTER_RESIZE_COEF_SCALE - ly;
        int hx = INTER_RESIZE_COEF_SCALE - lx;

        load_nv12_pixel<format>(luma, chroma, x_low,  y_low + ybatch_offset,  round_down2(x_low),  width, stride, r0[0], g0[0], b0[0]);
        load_nv12_pixel<format>(luma, chroma, x_high, y_low + ybatch_offset,  round_down2(x_high), width, stride, r0[1], g0[1], b0[1]);
        load_nv12_pixel<format>(luma, chroma, x_low,  y_high + ybatch_offset, round_down2(x_low),  width, stride, r0[2], g0[2], b0[2]);
        load_nv12_pixel<format>(luma, chroma, x_high, y_high + ybatch_offset, round_down2(x_high), width, stride, r0[3], g0[3], b0[3]);

        r = ( ((hy * ((hx * r0[0] + lx * r0[1]) >> 4)) >> 16) + ((ly * ((hx * r0[2] + lx * r0[3]) >> 4)) >> 16) + 2 )>>2;
        g = ( ((hy * ((hx * g0[0] + lx * g0[1]) >> 4)) >> 16) + ((ly * ((hx * g0[2] + lx * g0[3]) >> 4)) >> 16) + 2 )>>2;
        b = ( ((hy * ((hx * b0[0] + lx * b0[1]) >> 4)) >> 16) + ((ly * ((hx * b0[2] + lx * b0[3]) >> 4)) >> 16) + 2 )>>2;
    }
};

template<NV12Format nv12_format, typename OutDType, PixelLayout layout, Interpolation interp>
static __global__ void convert_nv12_to_rgb_kernel_4x(
    const void* luma, const void* chroma, 
    OutDType* pdst, float sx, float sy,
    int src_height, int src_width, int src_stride, float mean0, float mean1, float mean2, float scale0, float scale1, float scale2,
    int dst_width, int dst_stride, int dst_height, int nbatch
){
    int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= dst_width-3 || y >= dst_height) return;

    OutDType r[4], g[4], b[4];
    uint8_t r0, g0, b0;
    for(int ib = blockIdx.z; ib < nbatch; ib += gridDim.z){
        int ybatch_offset = ib * src_height;
        for(int ip = 0; ip < 4; ++ip){
            SamplePixel<OutDType, nv12_format, interp>::call(
                luma, chroma, x+ip, y, sx, sy, ybatch_offset, src_width, src_stride, src_height,
                r0, g0, b0
            );
            scale_rgb(r0, g0, b0, r[ip], g[ip], b[ip], mean0, mean1, mean2, scale0, scale1, scale2);
        }

        DataLayoutInvoker<OutDType, layout, Parallel::FourPixel>::call(
            pdst, r, g, b, ib, x, y, dst_width, dst_stride, dst_height
        );
    }
}

template<NV12Format nv12_format, typename OutDType, PixelLayout layout, Interpolation interp>
static __global__ void convert_nv12_to_rgb_kernel_1x(
    const void* luma, const void* chroma, 
    OutDType* pdst, float sx, float sy,
    int src_height, int src_width, int src_stride, float mean0, float mean1, float mean2, float scale0, float scale1, float scale2,
    int dst_width, int dst_stride, int dst_height, int nbatch
){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= dst_width || y >= dst_height) return;

    OutDType r, g, b;
    uint8_t r0, g0, b0;
    for(int ib = blockIdx.z; ib < nbatch; ib += gridDim.z){
        int ybatch_offset = ib * src_height;
        SamplePixel<OutDType, nv12_format, interp>::call(
            luma, chroma, x, y, sx, sy, ybatch_offset, src_width, src_stride, src_height,
            r0, g0, b0
        );

        scale_rgb(r0, g0, b0, r, g, b, mean0, mean1, mean2, scale0, scale1, scale2);
        DataLayoutInvoker<OutDType, layout, Parallel::SinglePixel>::call(
            pdst, r, g, b, ib, x, y, dst_width, dst_stride, dst_height
        );
    }
}

template<NV12Format nv12_format, DataType out_dtype, PixelLayout layout, Interpolation interp>
void batched_convert_nv12_to_rgb_impl(
    const void* luma, const void* chroma, int input_width, int input_stride, int input_height, int input_batch,
    void* out_ptr, int out_width, int out_stride, int out_height, 
    float mean0, float mean1, float mean2, float scale0, float scale1, float scale2,
    hipStream_t stream
){
    float sx = input_width  / (float)out_width;
    float sy = input_height / (float)out_height;
    using OutDType = typename AsPODType<out_dtype>::type;

    if(
        layout == PixelLayout::NHWC_BGR   || 
        layout == PixelLayout::NHWC_RGB   || 
        layout == PixelLayout::NCHW16_RGB || 
        layout == PixelLayout::NCHW16_BGR ||
        input_stride % 4 != 0
    ){
        int grid_z = input_batch >= 32 ? 32 : input_batch;
        dim3 dim_block(32, 32);
        dim3 dim_grid((out_width  + dim_block.x - 1) / dim_block.x,
                    (out_height + dim_block.y - 1) / dim_block.y, grid_z);
        convert_nv12_to_rgb_kernel_1x<nv12_format, OutDType, layout, interp> <<<dim_grid, dim_block, 0, stream>>>(
            luma, chroma, 
            (OutDType*)out_ptr, sx, sy, 
            input_height, input_width, input_stride,
            mean0, mean1, mean2, scale0, scale1, scale2, 
            out_width, out_stride, out_height, input_batch
        );

    }else{
        int grid_z = input_batch >= 32 ? 32 : input_batch;
        dim3 dim_block(16, 32);
        dim3 dim_grid(((out_width + 3) / 4  + dim_block.x - 1) / dim_block.x,
                       (out_height + dim_block.y - 1) / dim_block.y, grid_z);
        convert_nv12_to_rgb_kernel_4x<nv12_format, OutDType, layout, interp> <<<dim_grid, dim_block, 0, stream>>>(
            luma, chroma, 
            (OutDType*)out_ptr, sx, sy, 
            input_height, input_width, input_stride,
            mean0, mean1, mean2, scale0, scale1, scale2, 
            out_width, out_stride, out_height, input_batch
        );
    }
    checkRuntime(hipPeekAtLastError());
}

typedef void(*batched_convert_nv12_to_rgb_impl_function)(
    const void* luma, const void* chroma, int input_width, int input_stride, int input_height, int input_batch,
    void* out_ptr, int out_width, int out_stride, int out_height, 
    float mean0, float mean1, float mean2, float scale0, float scale1, float scale2,
    hipStream_t stream
);


// If you want to modify this part of the code, 
// please note that the order of the enumerated types must match the integer values of this type 
// (note: that the order starts from 1)

#define DefineNV12Format(...)                                               \
    batched_convert_nv12_to_rgb_impl<NV12Format::BlockLinear, __VA_ARGS__>,  \
    batched_convert_nv12_to_rgb_impl<NV12Format::PitchLinear, __VA_ARGS__>,

#define DefineDType(...)                                              \
    DefineNV12Format(DataType::Uint8, __VA_ARGS__)                    \
    DefineNV12Format(DataType::Float32, __VA_ARGS__)                  \
    DefineNV12Format(DataType::Float16, __VA_ARGS__)                  

#define DefineLayout(...)                                            \
    DefineDType(PixelLayout::NCHW_RGB, __VA_ARGS__)                  \
    DefineDType(PixelLayout::NCHW_BGR, __VA_ARGS__)                  \
    DefineDType(PixelLayout::NHWC_RGB, __VA_ARGS__)                  \
    DefineDType(PixelLayout::NHWC_BGR, __VA_ARGS__)                  \
    DefineDType(PixelLayout::NCHW16_RGB, __VA_ARGS__)                \
    DefineDType(PixelLayout::NCHW16_BGR, __VA_ARGS__)                  

#define DefineInterp                                          \
    DefineLayout(Interpolation::Nearest)                      \
    DefineLayout(Interpolation::Bilinear)              

#define DefineAllFunction   DefineInterp


template<typename T>struct EnumCount{};
template<> struct EnumCount<NV12Format>{static const unsigned int value = 2;};
template<> struct EnumCount<DataType>{static const unsigned int value = 3;};
template<> struct EnumCount<PixelLayout>{static const unsigned int value = 6;};
template<> struct EnumCount<Interpolation>{static const unsigned int value = 2;};

static const batched_convert_nv12_to_rgb_impl_function func_list[] = {
    DefineAllFunction
    nullptr
};  

void batched_convert_nv12_to_rgb(
    const void* luma, const void* chroma, int input_width, int input_stride, int input_height, int input_batch, NV12Format nv12_format, 
    void* out_ptr, int out_width, int out_stride, int out_height, DataType out_dtype, PixelLayout out_layout, Interpolation interp,
    float mean0, float mean1, float mean2, float scale0, float scale1, float scale2,
    void* stream
){
    int iformat = (int)nv12_format - 1;
    int odtype  = (int)out_dtype - 1;
    int olayout = (int)out_layout - 1;
    int iinterp  = (int)interp - 1;
    int index = ((iinterp * EnumCount<PixelLayout>::value + olayout) * EnumCount<DataType>::value + odtype) * EnumCount<NV12Format>::value + iformat;
    if(
        iformat < 0 || iformat >= EnumCount<NV12Format>::value ||
        odtype < 0  || odtype >= EnumCount<DataType>::value ||
        olayout < 0 || olayout >= EnumCount<PixelLayout>::value ||
        iinterp < 0 || iinterp >= EnumCount<Interpolation>::value ||
        index < 0 || index >= sizeof(func_list) / sizeof(func_list[0]) - 1
    ){
        fprintf(stderr, "Unsupported configure %d.\n", index);
        return;
    }

    batched_convert_nv12_to_rgb_impl_function func = func_list[index];
    func(
        luma, chroma, input_width, input_stride, input_height, input_batch,
        out_ptr, out_width, out_stride, out_height,
        mean0, mean1, mean2, scale0, scale1, scale2,
        (hipStream_t)stream
    );
}
